#include "hip/hip_runtime.h"
#include <assert.h>
#include "core.h"
#include "cuda_kernel.h"

void execute_kernel_compute_cuda(const Kernel &kernel, char *A, size_t bytes)
{
  assert(kernel.nb_blocks * kernel.threads_per_block * sizeof(double) == bytes);
  execute_kernel_compute_cuda_kernel<<<16, 256>>>(kernel.iterations, (double *)A);
}


__global__ void execute_kernel_compute_cuda_kernel(long iter, double *A)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  double A_local = A[tid];
  for (int i = 0; i < iter; i++) {
    A_local = fma(A_local, A_local, A_local);
  }
  A[tid] = A_local;
}
