#include "hip/hip_runtime.h"
#include <assert.h>
#include "core.h"
#include "cuda_kernel.h"

std::vector<char*> local_buffer;
size_t local_buffer_size;

#define UNROLL_1  1
#define UNROLL_4  4
#define UNROLL_8  8
#define UNROLL_16 16

#define CUDA_ERROR_CHECK(err)                                      \
  if (err != hipSuccess) {                                        \
    printf("CUDA calls error %s\n", hipGetErrorString(err));      \
  }                                                                \

__global__ void execute_kernel_compute_cuda_kernel_unroll_1(long iter, double *A);
__global__ void execute_kernel_compute_cuda_kernel_unroll_4(long iter, double *A);
__global__ void execute_kernel_compute_cuda_kernel_unroll_8(long iter, double *A);
__global__ void execute_kernel_compute_cuda_kernel_unroll_16(long iter, double *A);

void init_cuda_support(const std::vector<TaskGraph> &graphs)
{
  int nb_gpus = 1;
  hipError_t cuda_err;
  
  local_buffer.reserve(nb_gpus);
  int nb_blocks = graphs[0].kernel.nb_blocks;
  int threads_per_block = graphs[0].kernel.threads_per_block;
  int cuda_unroll = graphs[0].kernel.cuda_unroll;
  printf("init cuda support nb_blocks %d, threads_per_block %d, cuda_unroll %d\n", nb_blocks, threads_per_block, cuda_unroll);
  local_buffer_size = nb_blocks * threads_per_block * sizeof(double);
  for (int i = 0; i < nb_gpus; i++) {
    cuda_err = hipSetDevice(0);
    CUDA_ERROR_CHECK(cuda_err);
    cuda_err = hipMalloc((void**)&(local_buffer[i]), sizeof(double) * nb_blocks * threads_per_block * cuda_unroll);
    CUDA_ERROR_CHECK(cuda_err);
    assert(local_buffer[i] != NULL);
  }
}

void execute_kernel_compute_cuda(const Kernel &kernel, char *scratch_ptr, size_t scratch_bytes)
{
  hipError_t cuda_err;
  printf("CUDA COMPUTE KERNEL buffer %p, size %lld, nb_blocks %d, threads_per_block %d\n", scratch_ptr, scratch_bytes, kernel.nb_blocks, kernel.threads_per_block);
  assert(scratch_bytes <= local_buffer_size);
  if (kernel.memcpy_required == 1) {
    printf("enable memcpy in\n");
    cuda_err = hipMemcpy(local_buffer[kernel.gpu_id], scratch_ptr, scratch_bytes, hipMemcpyHostToDevice);
    CUDA_ERROR_CHECK(cuda_err);
  }
  assert(kernel.gpu_id == 0);
  cuda_err = hipSetDevice(kernel.gpu_id);
  CUDA_ERROR_CHECK(cuda_err);
  if (kernel.cuda_unroll == 4) {
    execute_kernel_compute_cuda_kernel_unroll_4<<<kernel.nb_blocks, kernel.threads_per_block>>>(kernel.iterations, (double *)local_buffer[kernel.gpu_id]);
  } else if (kernel.cuda_unroll == 8) {
    execute_kernel_compute_cuda_kernel_unroll_8<<<kernel.nb_blocks, kernel.threads_per_block>>>(kernel.iterations, (double *)local_buffer[kernel.gpu_id]);
  } else if (kernel.cuda_unroll == 16) {
    execute_kernel_compute_cuda_kernel_unroll_16<<<kernel.nb_blocks, kernel.threads_per_block>>>(kernel.iterations, (double *)local_buffer[kernel.gpu_id]);
  } else {
    execute_kernel_compute_cuda_kernel_unroll_1<<<kernel.nb_blocks, kernel.threads_per_block>>>(kernel.iterations, (double *)local_buffer[kernel.gpu_id]);
  }
  cuda_err = hipDeviceSynchronize();
  CUDA_ERROR_CHECK(cuda_err);
  if (kernel.memcpy_required == 1) {
    printf("enable memcpy out\n");
    cuda_err = hipMemcpy(scratch_ptr, local_buffer[kernel.gpu_id], scratch_bytes, hipMemcpyDeviceToHost);
    CUDA_ERROR_CHECK(cuda_err);
  }
}

__global__ void execute_kernel_compute_cuda_kernel_unroll_1(long iter, double *A)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  double A_local = A[tid];
  for (int i = 0; i < iter; i++) {
    A_local = fma(A_local, A_local, A_local);
  }
  A[tid] = A_local;
}

__global__ void execute_kernel_compute_cuda_kernel_unroll_4(long iter, double *A)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  double A_local[UNROLL_4];
  int u;
  #pragma unroll
  for (u = 0; u < UNROLL_4; u++) {
    A_local[u] = A[tid + u * blockDim.x * gridDim.x];
  }
  for (int i = 0; i < iter; i++) {
    #pragma unroll
    for (u = 0; u < UNROLL_4; u++) {
      A_local[u]= fma(A_local[u], A_local[u], A_local[u]);
    }
   // A_local = A_local * A_local + A_local;
  }
  #pragma unroll
  for (u = 0; u < UNROLL_4; u++) {
    A[tid + u * blockDim.x * gridDim.x] = A_local[u];
  }
}

__global__ void execute_kernel_compute_cuda_kernel_unroll_8(long iter, double *A)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  double A_local[UNROLL_8];
  int u;
  #pragma unroll
  for (u = 0; u < UNROLL_8; u++) {
    A_local[u] = A[tid + u * blockDim.x * gridDim.x];
  }
  for (int i = 0; i < iter; i++) {
    #pragma unroll
    for (u = 0; u < UNROLL_8; u++) {
      A_local[u]= fma(A_local[u], A_local[u], A_local[u]);
    }
   // A_local = A_local * A_local + A_local;
  }
  #pragma unroll
  for (u = 0; u < UNROLL_8; u++) {
    A[tid + u * blockDim.x * gridDim.x] = A_local[u];
  }
}

__global__ void execute_kernel_compute_cuda_kernel_unroll_16(long iter, double *A)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  double A_local[UNROLL_16];
  int u;
  #pragma unroll
  for (u = 0; u < UNROLL_16; u++) {
    A_local[u] = A[tid + u * blockDim.x * gridDim.x];
  }
  for (int i = 0; i < iter; i++) {
    #pragma unroll
    for (u = 0; u < UNROLL_16; u++) {
      A_local[u]= fma(A_local[u], A_local[u], A_local[u]);
    }
   // A_local = A_local * A_local + A_local;
  }
  #pragma unroll
  for (u = 0; u < UNROLL_16; u++) {
    A[tid + u * blockDim.x * gridDim.x] = A_local[u];
  }
}
