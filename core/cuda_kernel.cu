#include "hip/hip_runtime.h"
#include <assert.h>
#include "core.h"
#include "cuda_kernel.h"

std::vector<char*> local_buffer;
size_t local_buffer_size;

#define CUDA_ERROR_CHECK(err)                                      \
  if (err != hipSuccess) {                                        \
    printf("CUDA calls error %s\n", hipGetErrorString(err));      \
  }                                                                \

void init_cuda_support(const std::vector<TaskGraph> &graphs)
{
  int nb_gpus = 1;
  hipError_t cuda_err;
  
  local_buffer.reserve(nb_gpus);
  int nb_blocks = graphs[0].kernel.nb_blocks;
  int threads_per_block = graphs[0].kernel.threads_per_block;
  printf("init cuda support nb_blocks %d, threads_per_block %d\n", nb_blocks, threads_per_block);
  local_buffer_size = nb_blocks * threads_per_block * sizeof(double);
  for (int i = 0; i < nb_gpus; i++) {
    cuda_err = hipSetDevice(0);
    CUDA_ERROR_CHECK(cuda_err);
    cuda_err = hipMalloc((void**)&(local_buffer[i]), sizeof(double) * nb_blocks * threads_per_block);
    CUDA_ERROR_CHECK(cuda_err);
    assert(local_buffer[i] != NULL);
  }
}

void execute_kernel_compute_cuda(const Kernel &kernel, char *scratch_ptr, size_t scratch_bytes)
{
  hipError_t cuda_err;
  printf("CUDA COMPUTE KERNEL buffer %p, size %lld, nb_blocks %d, threads_per_block %d\n", scratch_ptr, scratch_bytes, kernel.nb_blocks, kernel.threads_per_block);
  assert(scratch_bytes <= local_buffer_size);
  if (kernel.memcpy_required == 1) {
    printf("enable memcpy in\n");
    cuda_err = hipMemcpy(local_buffer[kernel.gpu_id], scratch_ptr, scratch_bytes, hipMemcpyHostToDevice);
    CUDA_ERROR_CHECK(cuda_err);
  }
  assert(kernel.gpu_id == 0);
  cuda_err = hipSetDevice(kernel.gpu_id);
  CUDA_ERROR_CHECK(cuda_err);
  execute_kernel_compute_cuda_kernel<<<kernel.nb_blocks, kernel.threads_per_block>>>(kernel.iterations, (double *)local_buffer[kernel.gpu_id]);
  cuda_err = hipDeviceSynchronize();
  CUDA_ERROR_CHECK(cuda_err);
  if (kernel.memcpy_required == 1) {
    printf("enable memcpy out\n");
    cuda_err = hipMemcpy(scratch_ptr, local_buffer[kernel.gpu_id], scratch_bytes, hipMemcpyDeviceToHost);
    CUDA_ERROR_CHECK(cuda_err);
  }
}


__global__ void execute_kernel_compute_cuda_kernel(long iter, double *A)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  double A_local = A[tid];
  for (int i = 0; i < iter; i++) {
    A_local = fma(A_local, A_local, A_local);
  }
  A[tid] = A_local;
}
